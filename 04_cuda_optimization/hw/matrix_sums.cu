
#include <hip/hip_runtime.h>
#include <stdio.h>

// error checking macro
#define cudaCheckErrors(msg) \
    do { \
        hipError_t __err = hipGetLastError(); \
        if (__err != hipSuccess) { \
            fprintf(stderr, "Fatal error: %s (%s at %s:%d)\n", \
                msg, hipGetErrorString(__err), \
                __FILE__, __LINE__); \
            fprintf(stderr, "*** FAILED - ABORTING\n"); \
            exit(1); \
        } \
    } while (0)


const size_t DSIZE = 16384;
const int block_size = 256;

// matrix row-sum kernel
__global__ void row_sums(const float *A, float *sums, size_t ds) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < ds) {
        float sum = 0.0f;
        for (size_t i = 0; i < ds; ++i) {
            sum += A[idx * ds + i];
        }
        sums[idx] = sum;
    }
}

__global__ void column_sums(const float *A, float *sums, size_t ds) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < ds) {
        float sum = 0.0f;
        for (size_t i = 0; i < ds; ++i) {
            sum += A[i * ds + idx];
        }
        sums[idx] = sum;
    }
}

bool validate(float *data, size_t sz) {
    for (size_t i = 0; i < sz; ++i) {
        if (data[i] != (float)sz) {
            printf("Results mismatch at %lu, was: %f, expected: %f\n", i, data[i], (float)sz);
            return false;
        }
    }
    return true;
}

int main() {
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    float *h_A, *h_sums, *d_A, *d_sums;
    h_A = new float[DSIZE * DSIZE];
    h_sums = new float[DSIZE];

    for (int i = 0; i < DSIZE * DSIZE; ++i) {
        h_A[i] = 1.0f;
    }

    hipMalloc(&d_A, DSIZE * DSIZE * sizeof(float));
    hipMalloc(&d_sums, DSIZE * sizeof(float));
    cudaCheckErrors("hipMalloc failure");

    hipMemcpy(d_A, h_A, DSIZE * DSIZE * sizeof(float), hipMemcpyHostToDevice);
    cudaCheckErrors("hipMemcpy H2D fail");

    hipEventRecord(start, 0);
    row_sums<<<(DSIZE + block_size - 1) / block_size, block_size>>>(d_A, d_sums, DSIZE);
    cudaCheckErrors("kernel launch failure");
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);

    float elapsedTime;
    hipEventElapsedTime(&elapsedTime, start, stop);
    printf("row sum time: %f ms\n", elapsedTime);

    hipMemcpy(h_sums, d_sums, DSIZE * sizeof(float), hipMemcpyDeviceToHost);

    if (!validate(h_sums, DSIZE)) {
        printf("Validation failed\n");
        return -1;
    }
    printf("row sum correct\n");

    hipMemset(d_sums, 0, DSIZE * sizeof(float));

    hipEventRecord(start, 0);
    column_sums<<<(DSIZE + block_size - 1) / block_size, block_size>>>(d_A, d_sums, DSIZE);
    cudaCheckErrors("kernel launch failure");
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);

    hipEventElapsedTime(&elapsedTime, start, stop);
    printf("column sum time: %f ms\n", elapsedTime);

    hipMemcpy(h_sums, d_sums, DSIZE * sizeof(float), hipMemcpyDeviceToHost);

    if (!validate(h_sums, DSIZE)) {
        printf("Validation failed\n");
        return -1;
    }
    printf("column sum correct\n");

    hipFree(d_A);
    hipFree(d_sums);
    delete[] h_A;
    delete[] h_sums;

    return 0;
}