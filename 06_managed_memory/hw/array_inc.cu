
#include <hip/hip_runtime.h>
#include <cstdio>
#include <cstdlib>

// error checking macro
#define cudaCheckErrors(msg) \
    do { \
        hipError_t __err = hipGetLastError(); \
        if (__err != hipSuccess) { \
            fprintf(stderr, "Fatal error: %s (%s at %s:%d)\n", \
                msg, hipGetErrorString(__err), \
                __FILE__, __LINE__); \
            fprintf(stderr, "CUDA error code: %d\n", __err); \
            fprintf(stderr, "*** FAILED - ABORTING\n"); \
            exit(1); \
        } \
    } while (0)


template <typename T>
void alloc_bytes(T &ptr, size_t num_bytes) {
    ptr = (T)malloc(num_bytes);
}

__global__ void inc(int *array, size_t n) {
    size_t idx = threadIdx.x + blockDim.x * blockIdx.x;
    while (idx < n) {
        array[idx]++;
        idx += blockDim.x * gridDim.x; // grid-stride loop
    }
}

const size_t ds = 32ULL * 1024ULL * 1024ULL;

int main() {
    int *h_array, *d_array;

    alloc_bytes(h_array, ds * sizeof(h_array[0]));
    hipMalloc(&d_array, ds * sizeof(d_array[0]));
    cudaCheckErrors("hipMalloc Error");

    // No need to memset since you are going to overwrite the array with kernel results

    hipMemcpy(d_array, h_array, ds * sizeof(h_array[0]), hipMemcpyHostToDevice);
    cudaCheckErrors("hipMemcpy H->D Error");

    inc<<<256, 256>>>(d_array, ds);
    cudaCheckErrors("kernel launch error");

    hipMemcpy(h_array, d_array, ds * sizeof(h_array[0]), hipMemcpyDeviceToHost);
    cudaCheckErrors("kernel execution or hipMemcpy D->H Error");

    for (int i = 0; i < ds; i++) {
        if (h_array[i] != 1) {
            printf("mismatch at %d, was: %d, expected: %d\n", i, h_array[i], 1);
            return -1;
        }
    }

    printf("success!\n");

    // Don't forget to free the allocated memory
    free(h_array);
    hipFree(d_array);

    return 0;
}
