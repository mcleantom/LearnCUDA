
#include <hip/hip_runtime.h>
#include <stdio.h>

#define cudaCheckErrors(msg) \
    do { \
        hipError_t __err = hipGetLastError(); \
        if (__err != hipSuccess) { \
            fprintf(stderr, "Fatal error: %s (%s at %s:%d)\n", \
                msg, hipGetErrorString(__err), \
                __FILE__, __LINE__); \
            fprintf(stderr, "*** FAILED - ABORTING \n"); \
            exit(1); \
        } \
    } while(0)


const int DSIZE = 4096;
const int BLOCK_SIZE = 256;

__global__ void vadd(const float *A, const float *B, float *C, int ds) {
    int idx = threadIdx.x + blockIdx.x * threadIdx.x;
    if (idx < ds) {
        C[idx] = A[idx] + B[idx];
    }
}

int main() {
    float *h_A, *h_B, *h_C, *d_A, *d_B, *d_C;
    h_A = new float[DSIZE];
    h_B = new float[DSIZE];
    h_C = new float[DSIZE];
    for (int i=0; i < DSIZE; ++i) {
        h_A[i] = rand()/(float)RAND_MAX;
        h_B[i] = rand()/(float)RAND_MAX;
        h_C[i] = 0;
    }
    hipMalloc(&d_A, DSIZE*sizeof(float));
    hipMalloc(&d_B, DSIZE*sizeof(float));
    hipMalloc(&d_C, DSIZE*sizeof(float));

    cudaCheckErrors("hipMalloc failure");

    hipMemcpy(d_A, h_A, DSIZE*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, DSIZE*sizeof(float), hipMemcpyHostToDevice);
    cudaCheckErrors("hipMemcpy H2D failure");

    vadd<<<(DSIZE+BLOCK_SIZE-1)/BLOCK_SIZE, BLOCK_SIZE>>>(d_A, d_B, d_C, DSIZE);
    cudaCheckErrors("kernel launch failure");

    hipMemcpy(h_C, d_C, DSIZE*sizeof(float), hipMemcpyDeviceToHost);
    cudaCheckErrors("hipMemcpy D2H failure");

    printf("A[0] = %f\n", h_A[0]);
    printf("B[0] = %f\n", h_B[0]);
    printf("C[0] = %f\n", h_C[0]);

    free(h_A); free(h_B); free(h_C);
    hipFree(d_A); hipFree(h_B); hipFree(h_C);
    return 0;
}