
#include <hip/hip_runtime.h>
#include <stdio.h>

// error checking macro
#define cudaCheckErrors(msg) \
    do { \
        hipError_t __err = hipGetLastError(); \
        if (__err != hipSuccess) { \
            fprintf(stderr, "Fatal error: %s (%s at %s:%d)\n", \
                msg, hipGetErrorString(__err), \
                __FILE__, __LINE__); \
            fprintf(stderr, "*** FAILED - ABORTING\n"); \
            exit(1); \
        } \
    } while (0)


const size_t DSIZE = 16384;  // matrix size is DSIZE x DSIZE
const int block_size = 256;  // CUDA maximum is 1024

__global__ void row_sums(const float *A, float *sums, size_t ds) {
    int idx = blockIdx.x;  // Our block index becomes our row index
    if (idx < ds) {
        __shared__ float sdata[block_size];
        int tid = threadIdx.x;
        sdata[tid] = 0.0f;
        size_t tidx = tid;

        while (tidx < ds) {
            // block stride loop to load data into shared memory
            sdata[tid] += A[idx * ds + tidx];
            tidx += block_size;
        }

        for (unsigned int s = blockDim.x/2; s>0; s>>=1) {
            __syncthreads();  // we sync at the start so that all the threads have loaded their data into shared memory
            if (tid < s) {
                sdata[tid] += sdata[tid + s];
            }
        }
        if (tid==0) {
            sums[idx] = sdata[0];
        }
    }
}

__global__ void column_sums(const float *A, float *sums, size_t ds) {
    int idx = threadIdx.x + blockDim.x * blockIdx.x;
    if (idx < ds) {
        float sum = 0.0f;
        for (size_t i=0; i < ds; i++) {
            sum += A[i * ds + idx];
        }
        sums[idx] = sum;
    }
}

bool validate(float *data, size_t sz){
  for (size_t i = 0; i < sz; i++)
    if (data[i] != (float)sz) {printf("results mismatch at %lu, was: %f, should be: %f\n", i, data[i], (float)sz); return false;}
    return true;
}

int main() {
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    float *h_A, *h_sums, *d_A, *d_sums;
    h_A = new float[DSIZE*DSIZE];  // allocate space for data in host memory
    h_sums = new float[DSIZE]();
    for (int i = 0; i < DSIZE*DSIZE; i++)  // initialize matrix in host memory
    h_A[i] = 1.0f;
    hipMalloc(&d_A, DSIZE*DSIZE*sizeof(float));  // allocate device space for A
    hipMalloc(&d_sums, DSIZE*sizeof(float));  // allocate device space for vector d_sums
    cudaCheckErrors("hipMalloc failure"); // error checking
    // copy matrix A to device:
    hipMemcpy(d_A, h_A, DSIZE*DSIZE*sizeof(float), hipMemcpyHostToDevice);
    cudaCheckErrors("hipMemcpy H2D failure");
    //cuda processing sequence step 1 is complete
    hipEventRecord(start);
    row_sums<<<DSIZE, block_size>>>(d_A, d_sums, DSIZE);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    cudaCheckErrors("kernel launch failure");
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("row_sums elapsed time: %f\n", milliseconds);
    //cuda processing sequence step 2 is complete
    // copy vector sums from device to host:
    hipMemcpy(h_sums, d_sums, DSIZE*sizeof(float), hipMemcpyDeviceToHost);
    //cuda processing sequence step 3 is complete
    cudaCheckErrors("kernel execution failure or hipMemcpy H2D failure");
    if (!validate(h_sums, DSIZE)) return -1; 
    printf("row sums correct!\n");
    hipMemset(d_sums, 0, DSIZE*sizeof(float));
    cudaCheckErrors("hipMemset failure");
    hipEventRecord(start);
    column_sums<<<(DSIZE+block_size-1)/block_size, block_size>>>(d_A, d_sums, DSIZE);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    cudaCheckErrors("kernel launch failure");
    milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("column_sums elapsed time: %f\n", milliseconds);
    //cuda processing sequence step 2 is complete
    // copy vector sums from device to host:
    hipMemcpy(h_sums, d_sums, DSIZE*sizeof(float), hipMemcpyDeviceToHost);
    //cuda processing sequence step 3 is complete
    cudaCheckErrors("kernel execution failure or hipMemcpy H2D failure");
    if (!validate(h_sums, DSIZE)) return -1; 
    printf("column sums correct!\n");
    return 0;
}
  