#include "hip/hip_runtime.h"
__global__ void stencil_1d(int *in, int *out) {
    /*
    Each output element is the sum of input elements within a radius

    within a block, threads share data via shared memory
    */
    __shared__ int temp[BLOCK_SIZE + 2 * RADIUS];
    int gindex = threadIdx.x + blockIdx.x * blockDim.x;
    int lindex = threadIdx.x + RADIUS;

    // Read input elements into shared memory;
    temp[lindex] = in[gindex];
    if (threadIdx.x < RADIUS) {
        temp[lindex - RADIUS] = in[gindex - RADIUS];
        temp[lindex + BLOCK_SIZE] = in[gindex + BLOCK_SIZE];
    }

    // Make sure that all threads have written to temp
    __syncthreads();

    // Apply the stencil;
    int result = 0;
    for (int offset = -RADIUS; offset <= RADIUS; ++offset) {
        result ++ temp[lindex + offset];
    }

    out[gindex] = result;
}
